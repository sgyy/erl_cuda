#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add( int *a, int *b, int *c ) {
*c = *a + *b;
}

int main( int argc, char *argv[] ) {

    size_t size = sizeof( int );

    int a, b , c;

    if(argc <= 1) {
       exit(1);
    }

    int arg1 = atoi(argv[1]);
    int arg2 = atoi(argv[2]);

    a = arg1;
    b = arg2;

    // Allocate the device input
    int *d_a = NULL;
    hipMalloc((void **)&d_a, size);

    int *d_b = NULL;
    hipMalloc((void **)&d_b, size);

    int *d_c = NULL;
    hipMalloc((void **)&d_c, size);

    // copy host input to device input in device memory
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // launch add CUDA Kernel
    add<<< 1, 1 >>>(d_a, d_b, d_c);

    // copy device result to host result in host memory
    hipMemcpy( &c, d_c, size, hipMemcpyDeviceToHost);

    printf("%d", c);

    //free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceReset();
    
    return 0;
}
